#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <iostream>

using namespace std;

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_sample(int size = 1048576)
{   
    printf("UNIFIED SAMPLE:\n");
    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);
    hipMallocManaged(&c, nBytes);

    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    vectorAddGPU<<<grid, block>>>(a, b, c, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void pinned_sample(int size = 1048576)
{   
    printf("PINNED SAMPLE:\n");
    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    
    auto start1 = std::chrono::steady_clock::now();
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    hipHostRegister(a, nBytes, 0);
    hipHostRegister(b, nBytes, 0);
    hipHostRegister(c, nBytes, 0);
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start1;
    cout << "Pinned host malloc time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    start1 = std::chrono::steady_clock::now();
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end-start1;
    cout << "Pinned device malloc time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;

    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    start1 = std::chrono::steady_clock::now();
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end-start1;
    cout << "Pinned copy time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipHostUnregister(a);
    hipHostUnregister(b);
    hipHostUnregister(c);
    free(a);
    free(b);
    free(c);
}

void usual_sample(int size = 1048576)
{   
    printf("USUAL SAMPLE:\n");
    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    
    auto start1 = std::chrono::steady_clock::now();
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start1;
    cout << "Usual host malloc time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    
    start1 = std::chrono::steady_clock::now();
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end-start1;
    cout << "Usual device malloc time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    start1 = std::chrono::steady_clock::now();
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end-start1;
    cout << "Usual copy time: " << (elapsed_seconds.count())*1000.0 << " ms" << endl;
    
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}


int main(int argc, char **argv)
{
    int n = atoi(argv[1]);
    usual_sample(n);
    printf("\n");
    pinned_sample(n);
    printf("\n");
    unified_sample(n);
    
    return 0;
}
