
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipblas.h>

#define BLOCK_SIZE 32

__global__ void gpu_matrix_mult_global(float *a,
                                       float *b,
                                       float *c,
                                       int m,
                                       int n,
                                       int k)
{
    // TODO
}

__global__ void gpu_square_matrix_mult(float *d_a, float *d_b, float *d_result, int n)
{
    // TODO
}

void cpu_matrix_mult(float *h_a, float *h_b, float *h_result, int m, int n, int k)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            float tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int m, n, k;
    /* Fixed seed for illustration */
    srand(3333);
    m = atoi(argv[1]);
    n = atoi(argv[1]);
    k = atoi(argv[1]);
    
    // allocate memory in host RAM, h_cc is used to store CPU result
    float *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(float)*m*n, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(float)*n*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, sizeof(float)*m*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_cc, sizeof(float)*m*k, hipHostMallocDefault);
    
    // random initialize matrix A
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_a[i * n + j] = rand() % 1024;
        }
    }
    
    // random initialize matrix B
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            h_b[i * k + j] = rand() % 1024;
        }
    }
    
    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;
    
    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // start to count execution time of GPU version
    // Allocate memory space on the device
    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(float)*m*n);
    hipMalloc((void **) &d_b, sizeof(float)*n*k);
    hipMalloc((void **) &d_c, sizeof(float)*m*k);
    
    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*n*k, hipMemcpyHostToDevice);
    
    // TODO compute grid size
    
    hipEventRecord(start, 0);
    
    if(atoi(argv[2]) == 1)
    {
        // TODO call shared memory version
    }
    else if(atoi(argv[2]) == 2)
    {
        // TODO call global memory version
    }
    
    // Transefr results from device to host
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipMemcpy(h_c, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);
    
    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
    
    // start the CPU version
    hipEventRecord(start, 0);
    
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);
    
    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if(h_cc[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
    }
    
    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }
    
    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    
    return 0;
}
